
#include <hip/hip_runtime.h>
#include <stdio.h>

void setDevice(){
    // init value is 0
    int count_device = 0;
    hipError_t error = hipGetDeviceCount(&count_device);
    if (error != hipSuccess || count_device == 0) {
        printf("no CUDA capable devices were detected\n");
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        exit(-1);
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", count_device);
    }
    int set_device = 0;
    error = hipSetDevice(set_device);
    if (error != hipSuccess) {
        printf("hipSetDevice returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        exit(-1);
    }
    else
    {
        printf("Set device to %d\n", set_device);
    }
}